#include "hip/hip_runtime.h"
__global__ void matadd_kernel ( int* dest
                              , int* A, int* B
                              , std::size_t matWidth
                              , const int offsetX = 0, const int offsetY = 0
                              )
{
  const int posX = offsetX + (blockIdx.x * blockDim.x + threadIdx.x);
  const int posY = offsetY + (blockIdx.y * blockDim.y + threadIdx.y);

  const int id = posY * matWidth + posY;

  dest[id] = A[id] + B[id];
}

void matadd(int* hDest, int* hA, int* hB, std::size_t n)
{
  int maxThreads;
  hipDeviceGetAttribute(&maxThreads, hipDeviceAttributeMaxThreadsPerBlock, 0);

  std::cout << "Max " << maxThreads << " threads.\n\n";

  int threadDim = sqrt(maxThreads);
  
  int matsize = n * n * sizeof(int);

  int* dA, dB, dDest;

  hipMalloc(&dA, matsize);
  hipMalloc(&dB, matsize);
  hipMalloc(&dDest, matsize);

  hipMemcpy(dA, hA, matsize, hipMemcpyHostToDevice);
  hipMemcpy(dB, hB, matsize, hipMemcpyHostToDevice);

  dim3 blocks();
  dim3 threads();

  /* thinking.jpg */
  
  //  Coeur
  matadd_kernel<<< , >>> (dDest, dA, dB, n);
  
  //  Bas
  matadd_kernel<<< , >>> (dDest, dA, dB, n);

  //  Droite
  matadd_kernel<<< , >>> (dDest, dA, dB, n);

  //  Bas-droite
  matadd_kernel<<< , >>> (dDest, dA, dB, n);

  hipMemcpy(hDest, dDest, matsize, hipMemcpyDeviceToHost);
}